#include "hip/hip_runtime.h"
#include "extptr.h"
#include "types.h"

#include <fml/src/cpu/cpuvec.hh>

#include <fml/src/gpu/card.hh>
#include <fml/src/gpu/gpuhelpers.hh>
#include <fml/src/gpu/gpumat.hh>
#include <fml/src/gpu/linalg.hh>


// -----------------------------------------------------------------------------
// gpumat bindings
// -----------------------------------------------------------------------------

extern "C" SEXP R_gpumat_init(SEXP type, SEXP c_robj, SEXP m_, SEXP n_)
{
  SEXP ret;
  
  int m = INTEGER(m_)[0];
  int n = INTEGER(n_)[0];
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  
  #define FMLR_TMP_INIT(type) { \
    gpumat<type> *x = new gpumat<type>(*c); \
    x->resize(m, n); \
    newRptr(x, ret, fml_object_finalizer<gpumat<type>>); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_INIT(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_INIT(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_INIT(int)
  
  #undef FMLR_TMP_INIT
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_dim(SEXP type, SEXP x_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 2));
  
  #define FMLR_TMP_DIM(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    INTEGER(ret)[0] = x->nrows(); \
    INTEGER(ret)[1] = x->ncols(); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_DIM(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_DIM(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_DIM(int)
  
  #undef FMLR_TMP_DIM
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_inherit(SEXP x_robj, SEXP data)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  // TODO FIXME
  // x->inherit(REAL(data), LENGTH(data), false);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_resize(SEXP type, SEXP x_robj, SEXP m, SEXP n)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, resize, INTEGER(m)[0], INTEGER(n)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_print(SEXP type, SEXP x_robj, SEXP ndigits)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, print, INTEGER(ndigits)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_info(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, info);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_zero(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_zero);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_val(SEXP type, SEXP x_robj, SEXP v)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_val, DBL(v));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_linspace(SEXP type, SEXP x_robj, SEXP start, SEXP stop)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_linspace, DBL(start), DBL(stop));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_eye(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_eye);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_diag(SEXP type, SEXP x_robj, SEXP v_robj)
{
  #define FMLR_TMP_FILL_DIAG(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    x->fill_diag(*v); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_FILL_DIAG(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_FILL_DIAG(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_FILL_DIAG(int)
  
  #undef FMLR_TMP_FILL_DIAG
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_runif(SEXP type, SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  #define FMLR_TMP_FILL_RUNIF(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    if (INTEGER(seed)[0] == -1) \
      x->fill_runif((type) REAL(min)[0], (type) REAL(max)[0]); \
    else \
      x->fill_runif(INTEGER(seed)[0], (type) REAL(min)[0], (type) REAL(max)[0]); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_FILL_RUNIF(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_FILL_RUNIF(float)
  else
    error("unsupported fundamental type");
  
  #undef FMLR_TMP_FILL_RUNIF
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_rnorm(SEXP type, SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  #define FMLR_TMP_FILL_RNORM(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    if (INTEGER(seed)[0] == -1) \
      x->fill_rnorm((type) REAL(min)[0], (type) REAL(max)[0]); \
    else \
      x->fill_rnorm(INTEGER(seed)[0], (type) REAL(min)[0], (type) REAL(max)[0]); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_FILL_RNORM(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_FILL_RNORM(float)
  else
    error("unsupported fundamental type");
  
  #undef FMLR_TMP_FILL_RNORM
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_diag(SEXP type, SEXP x_robj, SEXP v_robj)
{
  #define FMLR_TMP_DIAG(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    x->diag(*v); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_DIAG(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_DIAG(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_DIAG(int)
  
  #undef FMLR_TMP_DIAG
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_antidiag(SEXP type, SEXP x_robj, SEXP v_robj)
{
  #define FMLR_TMP_ANTIDIAG(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    x->antidiag(*v); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_ANTIDIAG(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_ANTIDIAG(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_ANTIDIAG(int)
  
  #undef FMLR_TMP_ANTIDIAG
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_scale(SEXP type, SEXP x_robj, SEXP s)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, scale, DBL(s));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_rev_rows(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, rev_rows);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_rev_cols(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, rev_cols);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_to_robj(SEXP type, SEXP x_robj)
{
  SEXP ret;
  
  if (INT(type) == TYPE_DOUBLE)
  {
    gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(REALSXP, m, n));
    cpumat<double> ret_mat(REAL(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  else if (INT(type) == TYPE_FLOAT)
  {
    gpumat<float> *x = (gpumat<float>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(INTSXP, m, n));
    cpumat<float> ret_mat((float*) INTEGER(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  else //if (INT(type) == TYPE_INT)
  {
    gpumat<int> *x = (gpumat<int>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(INTSXP, m, n));
    cpumat<int> ret_mat(INTEGER(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_from_robj(SEXP x_robj, SEXP robj)
{
  int m = nrows(robj);
  int n = ncols(robj);
  
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  len_t m_x = x->nrows();
  len_t n_x = x->ncols();
  
  if (m_x != m || n_x != n)
    x->resize(m, n);
  
  cpumat<double> robj_mat(REAL(robj), m, n, false);
  gpuhelpers::cpu2gpu(robj_mat, *x);
  
  return R_NilValue;
}



// -----------------------------------------------------------------------------
// linalg namespace
// -----------------------------------------------------------------------------

template <typename REAL>
static inline void crossprod(bool xpose, REAL alpha, void *x, void *ret)
{
  CAST_MAT(gpumat, REAL, x_cast, x);
  CAST_MAT(gpumat, REAL, ret_cast, ret);
  if (xpose)
    linalg::tcrossprod(alpha, *x_cast, *ret_cast);
  else
    linalg::crossprod(alpha, *x_cast, *ret_cast);
}

extern "C" SEXP R_gpumat_linalg_crossprod(SEXP type, SEXP xpose, SEXP alpha, SEXP x_robj, SEXP ret_robj)
{
  void *x = getRptr(x_robj);
  void *ret = getRptr(ret_robj);
  APPLY_TEMPLATED_FUNCTION(type, crossprod, LGL(xpose), DBL(alpha), x, ret);
  
  return R_NilValue;
}
