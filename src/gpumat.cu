#include "hip/hip_runtime.h"
#define R_USE_C99_IN_CXX
#define FML_PRINT_R

#include "apply.hpp"
#include "extptr.hpp"
#include "types.h"

#include <fml/src/cpu/cpuvec.hh>

#include <fml/src/gpu/card.hh>
#include <fml/src/gpu/gpuhelpers.hh>
#include <fml/src/gpu/gpumat.hh>


extern "C" SEXP R_gpumat_init(SEXP type, SEXP c_robj, SEXP m_, SEXP n_)
{
  SEXP ret;
  
  int m = INTEGER(m_)[0];
  int n = INTEGER(n_)[0];
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  
  #define FMLR_TMP_INIT(type) { \
    gpumat<type> *x = new gpumat<type>(*c); \
    TRY_CATCH( x->resize(m, n) ) \
    newRptr(x, ret, fml_object_finalizer<gpumat<type>>); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_INIT(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_INIT(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_INIT(int)
  
  #undef FMLR_TMP_INIT
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_dim(SEXP type, SEXP x_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 2));
  
  #define FMLR_TMP_DIM(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    INTEGER(ret)[0] = x->nrows(); \
    INTEGER(ret)[1] = x->ncols(); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_DIM(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_DIM(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_DIM(int)
  
  #undef FMLR_TMP_DIM
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_resize(SEXP type, SEXP x_robj, SEXP m, SEXP n)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, resize, INTEGER(m)[0], INTEGER(n)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_dupe(SEXP type, SEXP x_robj)
{
  SEXP ret;
  
  #define FMLR_TMP_DUPE(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpumat<type> *y = new gpumat<type>(x->get_card()); \
    gpuhelpers::gpu2gpu(*x, *y); \
    newRptr(y, ret, fml_object_finalizer<gpumat<type>>); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_DUPE(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_DUPE(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_DUPE(int)
  
  #undef FMLR_TMP_DUPE
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_print(SEXP type, SEXP x_robj, SEXP ndigits)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, print, INTEGER(ndigits)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_info(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, info);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_zero(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_zero);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_val(SEXP type, SEXP x_robj, SEXP v)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_val, DBL(v));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_linspace(SEXP type, SEXP x_robj, SEXP start, SEXP stop)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_linspace, DBL(start), DBL(stop));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_eye(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_eye);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_diag(SEXP type, SEXP x_robj, SEXP v_robj)
{
  #define FMLR_TMP_FILL_DIAG(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    x->fill_diag(*v); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_FILL_DIAG(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_FILL_DIAG(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_FILL_DIAG(int)
  
  #undef FMLR_TMP_FILL_DIAG
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_runif(SEXP type, SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  #define FMLR_TMP_FILL_RUNIF(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    if (INTEGER(seed)[0] == -1) \
      x->fill_runif((type) REAL(min)[0], (type) REAL(max)[0]); \
    else \
      x->fill_runif(INTEGER(seed)[0], (type) REAL(min)[0], (type) REAL(max)[0]); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_FILL_RUNIF(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_FILL_RUNIF(float)
  else
    error(TYPE_ERR);
  
  #undef FMLR_TMP_FILL_RUNIF
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_rnorm(SEXP type, SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  #define FMLR_TMP_FILL_RNORM(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    if (INTEGER(seed)[0] == -1) \
      x->fill_rnorm((type) REAL(min)[0], (type) REAL(max)[0]); \
    else \
      x->fill_rnorm(INTEGER(seed)[0], (type) REAL(min)[0], (type) REAL(max)[0]); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_FILL_RNORM(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_FILL_RNORM(float)
  else
    error(TYPE_ERR);
  
  #undef FMLR_TMP_FILL_RNORM
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_diag(SEXP type, SEXP x_robj, SEXP v_robj)
{
  #define FMLR_TMP_DIAG(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    x->diag(*v); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_DIAG(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_DIAG(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_DIAG(int)
  
  #undef FMLR_TMP_DIAG
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_antidiag(SEXP type, SEXP x_robj, SEXP v_robj)
{
  #define FMLR_TMP_ANTIDIAG(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    x->antidiag(*v); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_ANTIDIAG(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_ANTIDIAG(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_ANTIDIAG(int)
  
  #undef FMLR_TMP_ANTIDIAG
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_scale(SEXP type, SEXP x_robj, SEXP s)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, scale, DBL(s));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_rev_rows(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, rev_rows);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_rev_cols(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, rev_cols);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_get(SEXP type, SEXP x_robj, SEXP i, SEXP j)
{
  SEXP ret;
  PROTECT(ret = allocVector(REALSXP, 1));
  
  #define FMLR_TMP_GET(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    TRY_CATCH( DBL(ret) = (double) x->get(INT(i), INT(j)) ) }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_GET(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_GET(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_GET(int)
  
  #undef FMLR_TMP_GET
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_set(SEXP type, SEXP x_robj, SEXP i, SEXP j, SEXP v)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, set, INT(i), INT(j), DBL(v));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_get_row(SEXP type, SEXP x_robj, SEXP i, SEXP v_robj)
{
  #define FMLR_TMP_GET_ROW(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    TRY_CATCH( x->get_row(INT(i), *v) ) }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_GET_ROW(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_GET_ROW(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_GET_ROW(int)
  
  #undef FMLR_TMP_GET_ROW
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_get_col(SEXP type, SEXP x_robj, SEXP j, SEXP v_robj)
{
  #define FMLR_TMP_GET_COL(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *v = (gpuvec<type>*) getRptr(v_robj); \
    TRY_CATCH( x->get_col(INT(j), *v) ) }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_GET_COL(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_GET_COL(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_GET_COL(int)
  
  #undef FMLR_TMP_GET_COL
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_to_robj(SEXP type, SEXP x_robj)
{
  SEXP ret;
  
  if (INT(type) == TYPE_DOUBLE)
  {
    gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(REALSXP, m, n));
    cpumat<double> ret_mat(REAL(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  else if (INT(type) == TYPE_FLOAT)
  {
    gpumat<float> *x = (gpumat<float>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(INTSXP, m, n));
    cpumat<float> ret_mat((float*) INTEGER(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  else //if (INT(type) == TYPE_INT)
  {
    gpumat<int> *x = (gpumat<int>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(INTSXP, m, n));
    cpumat<int> ret_mat(INTEGER(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_from_robj(SEXP type, SEXP x_robj, SEXP type_robj, SEXP robj)
{
  const int m = nrows(robj);
  const int n = ncols(robj);
  
  #define FML_TMP_MATCOPY(type_robj) \
    if (x->nrows() != m || x->ncols() != n) \
      x->resize(m, n); \
    if (INT(type_robj) == TYPE_DOUBLE) \
    { \
      cpumat<double> robj_mat(REAL(robj), m, n, false); \
      TRY_CATCH( gpuhelpers::cpu2gpu(robj_mat, *x) ) \
    } \
    else if (INT(type_robj) == TYPE_FLOAT) \
    { \
      cpumat<float> robj_mat(FLOAT(robj), m, n, false); \
      TRY_CATCH( gpuhelpers::cpu2gpu(robj_mat, *x) ) \
    } \
    else \
    { \
      cpumat<int> robj_mat(INTEGER(robj), m, n, false); \
      TRY_CATCH( gpuhelpers::cpu2gpu(robj_mat, *x) ) \
    }
  
  if (INT(type) == TYPE_DOUBLE)
  {
    CAST_FML(gpumat, double, x, x_robj);
    FML_TMP_MATCOPY(type_robj)
  }
  else if (INT(type) == TYPE_FLOAT)
  {
    CAST_FML(gpumat, float, x, x_robj);
    FML_TMP_MATCOPY(type_robj)
  }
  else if (INT(type) == TYPE_INT)
  {
    CAST_FML(gpumat, int, x, x_robj);
    FML_TMP_MATCOPY(type_robj)
  }
  
  #undef FML_TMP_MATCOPY
  
  return R_NilValue;
}
