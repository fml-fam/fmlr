#include "hip/hip_runtime.h"
#include "extptr.h"

#include "fml/src/cpu/cpuvec.hh"

#include "fml/src/gpu/card.hh"
#include "fml/src/gpu/gpuhelpers.hh"
#include "fml/src/gpu/gpumat.hh"
#include "fml/src/gpu/linalg.hh"


// -----------------------------------------------------------------------------
// gpumat bindings
// -----------------------------------------------------------------------------

extern "C" SEXP R_gpumat_init(SEXP c_robj, SEXP m_, SEXP n_)
{
  SEXP ret;
  
  int m = INTEGER(m_)[0];
  int n = INTEGER(n_)[0];
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  
  gpumat<double> *x = new gpumat<double>(*c);
  if (m > 0 && n > 0)
    x->resize(m, n);
  
  newRptr(x, ret, fml_object_finalizer<gpumat<double>>);
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_dim(SEXP x_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 2));
  
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  INTEGER(ret)[0] = x->nrows();
  INTEGER(ret)[1] = x->ncols();
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_inherit(SEXP x_robj, SEXP data)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  // TODO FIXME
  // x->inherit(REAL(data), LENGTH(data), false);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_resize(SEXP x_robj, SEXP m, SEXP n)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->resize(INTEGER(m)[0], INTEGER(n)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_print(SEXP x_robj, SEXP ndigits)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->print(INTEGER(ndigits)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_info(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->info();
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_zero(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_zero();
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_val(SEXP x_robj, SEXP v)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_val(REAL(v)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_linspace(SEXP x_robj, SEXP start, SEXP stop)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_linspace(REAL(start)[0], REAL(stop)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_eye(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_eye();
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_runif(SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  if (INTEGER(seed)[0] == -1)
    x->fill_runif(REAL(min)[0], REAL(max)[0]);
  else
    x->fill_runif(INTEGER(seed)[0], REAL(min)[0], REAL(max)[0]);
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_rnorm(SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  if (INTEGER(seed)[0] == -1)
    x->fill_rnorm(REAL(min)[0], REAL(max)[0]);
  else
    x->fill_rnorm(INTEGER(seed)[0], REAL(min)[0], REAL(max)[0]);
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_scale(SEXP x_robj, SEXP s)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->scale(REAL(s)[0]);
  return R_NilValue;
}



// extern "C" SEXP R_gpumat_rev_rows(SEXP x_robj)
// {
//   gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
//   x->rev_rows();
//   return R_NilValue;
// }



// extern "C" SEXP R_gpumat_rev_cols(SEXP x_robj)
// {
//   gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
//   x->rev_cols();
//   return R_NilValue;
// }



extern "C" SEXP R_gpumat_to_robj(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  len_t m = x->nrows();
  len_t n = x->ncols();
  
  SEXP ret;
  PROTECT(ret = allocMatrix(REALSXP, m, n));
  
  cpumat<double> ret_mat(REAL(ret), m, n, false);
  gpuhelpers::gpu2cpu(*x, ret_mat);
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_from_robj(SEXP x_robj, SEXP robj)
{
  int m = nrows(robj);
  int n = ncols(robj);
  
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  len_t m_x = x->nrows();
  len_t n_x = x->ncols();
  
  if (m_x != m || n_x != n)
    x->resize(m, n);
  
  cpumat<double> robj_mat(REAL(robj), m, n, false);
  gpuhelpers::cpu2gpu(robj_mat, *x);
  
  return R_NilValue;
}



// -----------------------------------------------------------------------------
// linalg namespace
// -----------------------------------------------------------------------------

extern "C" SEXP R_gpumat_linalg_crossprod(SEXP xpose, SEXP alpha, SEXP x_robj, SEXP ret_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  gpumat<double> *ret = (gpumat<double>*) getRptr(ret_robj);
  
  if (LOGICAL(xpose)[0])
    linalg::tcrossprod(REAL(alpha)[0], *x, *ret);
  else
    linalg::crossprod(REAL(alpha)[0], *x, *ret);
  
  return R_NilValue;
}
