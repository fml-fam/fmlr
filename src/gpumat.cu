#include "hip/hip_runtime.h"
#include "extptr.h"
#include "types.h"

#include "fml/src/cpu/cpuvec.hh"

#include "fml/src/gpu/card.hh"
#include "fml/src/gpu/gpuhelpers.hh"
#include "fml/src/gpu/gpumat.hh"
#include "fml/src/gpu/linalg.hh"


// -----------------------------------------------------------------------------
// gpumat bindings
// -----------------------------------------------------------------------------

extern "C" SEXP R_gpumat_init(SEXP type, SEXP c_robj, SEXP m_, SEXP n_)
{
  SEXP ret;
  
  int m = INTEGER(m_)[0];
  int n = INTEGER(n_)[0];
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  
  if (INT(type) == TYPE_DOUBLE)
  {
    gpumat<double> *x = new gpumat<double>(*c);
    x->resize(m, n);
    newRptr(x, ret, fml_object_finalizer<gpumat<double>>);
  }
  else if (INT(type) == TYPE_FLOAT)
  {
    gpumat<float> *x = new gpumat<float>(*c);
    x->resize(m, n);
    newRptr(x, ret, fml_object_finalizer<gpumat<float>>);
  }
  else //if (INT(type) == TYPE_INT)
  {
    gpumat<int> *x = new gpumat<int>(*c);
    x->resize(m, n);
    newRptr(x, ret, fml_object_finalizer<gpumat<int>>);
  }
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_dim(SEXP type, SEXP x_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 2));
  
  if (INT(type) == TYPE_DOUBLE)
  {
    gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
    INTEGER(ret)[0] = x->nrows();
    INTEGER(ret)[1] = x->ncols();
  }
  else if (INT(type) == TYPE_FLOAT)
  {
    gpumat<float> *x = (gpumat<float>*) getRptr(x_robj);
    INTEGER(ret)[0] = x->nrows();
    INTEGER(ret)[1] = x->ncols();
  }
  else //if (INT(type) == TYPE_INT)
  {
    gpumat<int> *x = (gpumat<int>*) getRptr(x_robj);
    INTEGER(ret)[0] = x->nrows();
    INTEGER(ret)[1] = x->ncols();
  }
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_inherit(SEXP x_robj, SEXP data)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  // TODO FIXME
  // x->inherit(REAL(data), LENGTH(data), false);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_resize(SEXP type, SEXP x_robj, SEXP m, SEXP n)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, resize, INTEGER(m)[0], INTEGER(n)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_print(SEXP type, SEXP x_robj, SEXP ndigits)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, print, INTEGER(ndigits)[0]);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_info(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, info);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_zero(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_zero);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_val(SEXP type, SEXP x_robj, SEXP v)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_val, DBL(v));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_linspace(SEXP type, SEXP x_robj, SEXP start, SEXP stop)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_linspace, DBL(start), DBL(stop));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_eye(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, fill_eye);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_runif(SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  if (INTEGER(seed)[0] == -1)
    x->fill_runif(REAL(min)[0], REAL(max)[0]);
  else
    x->fill_runif(INTEGER(seed)[0], REAL(min)[0], REAL(max)[0]);
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_fill_rnorm(SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  if (INTEGER(seed)[0] == -1)
    x->fill_rnorm(REAL(min)[0], REAL(max)[0]);
  else
    x->fill_rnorm(INTEGER(seed)[0], REAL(min)[0], REAL(max)[0]);
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_scale(SEXP type, SEXP x_robj, SEXP s)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, scale, DBL(s));
  return R_NilValue;
}



extern "C" SEXP R_gpumat_rev_rows(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, rev_rows);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_rev_cols(SEXP type, SEXP x_robj)
{
  APPLY_TEMPLATED_METHOD(gpumat, type, x_robj, rev_cols);
  return R_NilValue;
}



extern "C" SEXP R_gpumat_to_robj(SEXP type, SEXP x_robj)
{
  SEXP ret;
  
  if (INT(type) == TYPE_DOUBLE)
  {
    gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(REALSXP, m, n));
    cpumat<double> ret_mat(REAL(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  else if (INT(type) == TYPE_FLOAT)
  {
    gpumat<float> *x = (gpumat<float>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(INTSXP, m, n));
    cpumat<float> ret_mat((float*) INTEGER(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  else //if (INT(type) == TYPE_INT)
  {
    gpumat<int> *x = (gpumat<int>*) getRptr(x_robj);
    len_t m = x->nrows();
    len_t n = x->ncols();
    
    PROTECT(ret = allocMatrix(INTSXP, m, n));
    cpumat<int> ret_mat(INTEGER(ret), m, n, false);
    gpuhelpers::gpu2cpu(*x, ret_mat);
  }
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_from_robj(SEXP x_robj, SEXP robj)
{
  int m = nrows(robj);
  int n = ncols(robj);
  
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  len_t m_x = x->nrows();
  len_t n_x = x->ncols();
  
  if (m_x != m || n_x != n)
    x->resize(m, n);
  
  cpumat<double> robj_mat(REAL(robj), m, n, false);
  gpuhelpers::cpu2gpu(robj_mat, *x);
  
  return R_NilValue;
}



// -----------------------------------------------------------------------------
// linalg namespace
// -----------------------------------------------------------------------------

template <typename REAL>
static inline void crossprod(bool xpose, REAL alpha, void *x, void *ret)
{
  CAST_MAT(gpumat, REAL, x_cast, x);
  CAST_MAT(gpumat, REAL, ret_cast, ret);
  if (xpose)
    linalg::tcrossprod(alpha, *x_cast, *ret_cast);
  else
    linalg::crossprod(alpha, *x_cast, *ret_cast);
}

extern "C" SEXP R_gpumat_linalg_crossprod(SEXP type, SEXP xpose, SEXP alpha, SEXP x_robj, SEXP ret_robj)
{
  void *x = getRptr(x_robj);
  void *ret = getRptr(ret_robj);
  APPLY_TEMPLATED_FUNCTION(type, crossprod, LGL(xpose), DBL(alpha), x, ret);
  
  return R_NilValue;
}
