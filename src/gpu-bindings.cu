#include "hip/hip_runtime.h"
#include "extptr.h"

#include "fml/src/cpu/cpuvec.hh"

#include "fml/src/gpu/card.hh"
#include "fml/src/gpu/gpuhelpers.hh"
#include "fml/src/gpu/gpuvec.hh"
#include "fml/src/gpu/linalg.hh"


// -----------------------------------------------------------------------------
// card bindings
// -----------------------------------------------------------------------------

extern "C" SEXP R_card_init(SEXP id_)
{
  SEXP ret;
  
  int id = INTEGER(id_)[0];
  card *c = new card(id);
  std::shared_ptr<card> *x = new std::shared_ptr<card>(c);
  
  newRptr(x, ret, fml_object_finalizer<std::shared_ptr<card>>);
  UNPROTECT(1);
  return ret;
}

extern "C" SEXP R_card_set(SEXP c_robj, SEXP id)
{
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  (*c)->set(INTEGER(id)[0]);
  
  return R_NilValue;
}

extern "C" SEXP R_card_info(SEXP c_robj)
{
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  (*c)->info();
  
  return R_NilValue;
}

extern "C" SEXP R_card_get_id(SEXP c_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 1));
  
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  INTEGER(ret)[0] = (*c)->get_id();
  UNPROTECT(1);
  return ret;
}

extern "C" SEXP R_card_valid_card(SEXP c_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(LGLSXP, 1));
  
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  LOGICAL(ret)[0] = (*c)->valid_card();
  UNPROTECT(1);
  return ret;
}



// -----------------------------------------------------------------------------
// gpuvec bindings
// -----------------------------------------------------------------------------

extern "C" SEXP R_gpuvec_init(SEXP c_robj, SEXP size_)
{
  SEXP ret;
  
  int size = INTEGER(size_)[0];
  
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  
  gpuvec<double> *x = new gpuvec<double>(*c);
  if (size > 0)
    x->resize(size);
  
  newRptr(x, ret, fml_object_finalizer<gpuvec<double>>);
  UNPROTECT(1);
  return ret;
}

extern "C" SEXP R_gpuvec_size(SEXP x_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 1));
  
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  INTEGER(ret)[0] = x->size();
  UNPROTECT(1);
  return ret;
}

extern "C" SEXP R_gpuvec_set(SEXP x_robj, SEXP data)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  // TODO FIXME
  // x->set(REAL(data), LENGTH(data), false);
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_resize(SEXP x_robj, SEXP size)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->resize(INTEGER(size)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_print(SEXP x_robj, SEXP ndigits)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->print(INTEGER(ndigits)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_info(SEXP x_robj)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->info();
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_fill_zero(SEXP x_robj)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->fill_zero();
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_fill_one(SEXP x_robj)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->fill_one();
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_fill_val(SEXP x_robj, SEXP v)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->fill_val(REAL(v)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_fill_linspace(SEXP x_robj, SEXP start, SEXP stop)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->fill_linspace(REAL(start)[0], REAL(stop)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_scale(SEXP x_robj, SEXP s)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->scale(REAL(s)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_rev(SEXP x_robj)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  x->rev();
  return R_NilValue;
}

extern "C" SEXP R_gpuvec_to_robj(SEXP x_robj)
{
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  len_t size = x->size();
  
  SEXP ret;
  PROTECT(ret = allocVector(REALSXP, size));
  
  cpuvec<double> ret_vec(REAL(ret), size, false);
  gpuhelpers::gpu2cpu(*x, ret_vec);
  
  UNPROTECT(1);
  return ret;
}

extern "C" SEXP R_gpuvec_from_robj(SEXP x_robj, SEXP robj)
{
  int size = LENGTH(robj);
  
  gpuvec<double> *x = (gpuvec<double>*) getRptr(x_robj);
  len_t size_x = x->size();
  
  if (size_x != size)
    x->resize(size);
  
  cpuvec<double> robj_vec(REAL(robj), size, false);
  gpuhelpers::cpu2gpu(robj_vec, *x);
  
  return R_NilValue;
}



// -----------------------------------------------------------------------------
// gpumat bindings
// -----------------------------------------------------------------------------

extern "C" SEXP R_gpumat_init(SEXP c_robj, SEXP m_, SEXP n_)
{
  SEXP ret;
  
  int m = INTEGER(m_)[0];
  int n = INTEGER(n_)[0];
  std::shared_ptr<card> *c = (std::shared_ptr<card>*) getRptr(c_robj);
  
  gpumat<double> *x = new gpumat<double>(*c);
  if (m > 0 && n > 0)
    x->resize(m, n);
  
  newRptr(x, ret, fml_object_finalizer<gpumat<double>>);
  UNPROTECT(1);
  return ret;
}

extern "C" SEXP R_gpumat_dim(SEXP x_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(INTSXP, 2));
  
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  INTEGER(ret)[0] = x->nrows();
  INTEGER(ret)[1] = x->ncols();
  
  return ret;
}

extern "C" SEXP R_gpumat_set(SEXP x_robj, SEXP data)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  // TODO FIXME
  // x->set(REAL(data), LENGTH(data), false);
  return R_NilValue;
}

extern "C" SEXP R_gpumat_resize(SEXP x_robj, SEXP m, SEXP n)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->resize(INTEGER(m)[0], INTEGER(n)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpumat_print(SEXP x_robj, SEXP ndigits)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->print(INTEGER(ndigits)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpumat_info(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->info();
  return R_NilValue;
}

extern "C" SEXP R_gpumat_fill_zero(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_zero();
  return R_NilValue;
}

extern "C" SEXP R_gpumat_fill_one(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_one();
  return R_NilValue;
}

extern "C" SEXP R_gpumat_fill_val(SEXP x_robj, SEXP v)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_val(REAL(v)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpumat_fill_linspace(SEXP x_robj, SEXP start, SEXP stop)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_linspace(REAL(start)[0], REAL(stop)[0]);
  return R_NilValue;
}

extern "C" SEXP R_gpumat_fill_eye(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->fill_eye();
  return R_NilValue;
}

// TODO diag

extern "C" SEXP R_gpumat_fill_runif(SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  if (INTEGER(seed)[0] == -1)
    x->fill_runif(REAL(min)[0], REAL(max)[0]);
  else
    x->fill_runif(INTEGER(seed)[0], REAL(min)[0], REAL(max)[0]);
  
  return R_NilValue;
}

extern "C" SEXP R_gpumat_fill_rnorm(SEXP x_robj, SEXP seed, SEXP min, SEXP max)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  if (INTEGER(seed)[0] == -1)
    x->fill_rnorm(REAL(min)[0], REAL(max)[0]);
  else
    x->fill_rnorm(INTEGER(seed)[0], REAL(min)[0], REAL(max)[0]);
  
  return R_NilValue;
}

extern "C" SEXP R_gpumat_scale(SEXP x_robj, SEXP s)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  x->scale(REAL(s)[0]);
  return R_NilValue;
}

// extern "C" SEXP R_gpumat_rev_rows(SEXP x_robj)
// {
//   gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
//   x->rev_rows();
//   return R_NilValue;
// }
// 
// extern "C" SEXP R_gpumat_rev_cols(SEXP x_robj)
// {
//   gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
//   x->rev_cols();
//   return R_NilValue;
// }

extern "C" SEXP R_gpumat_to_robj(SEXP x_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  len_t m = x->nrows();
  len_t n = x->ncols();
  
  SEXP ret;
  PROTECT(ret = allocMatrix(REALSXP, m, n));
  
  cpumat<double> ret_mat(REAL(ret), m, n, false);
  gpuhelpers::gpu2cpu(*x, ret_mat);
  
  UNPROTECT(1);
  return ret;
}

extern "C" SEXP R_gpumat_from_robj(SEXP x_robj, SEXP robj)
{
  int m = nrows(robj);
  int n = ncols(robj);
  
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  len_t m_x = x->nrows();
  len_t n_x = x->ncols();
  
  if (m_x != m || n_x != n)
    x->resize(m, n);
  
  cpumat<double> robj_mat(REAL(robj), m, n, false);
  gpuhelpers::cpu2gpu(robj_mat, *x);
  
  return R_NilValue;
}



// -----------------------------------------------------------------------------
// linalg namespace
// -----------------------------------------------------------------------------

extern "C" SEXP R_gpumat_linalg_crossprod(SEXP xpose, SEXP alpha, SEXP x_robj, SEXP ret_robj)
{
  gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
  gpumat<double> *ret = (gpumat<double>*) getRptr(ret_robj);
  
  if (LOGICAL(xpose)[0])
    linalg::tcrossprod(REAL(alpha)[0], *x, *ret);
  else
    linalg::crossprod(REAL(alpha)[0], *x, *ret);
  
  return R_NilValue;
}
