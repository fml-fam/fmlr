#include "hip/hip_runtime.h"
#include "apply.hpp"
#include "extptr.hpp"
#include "types.h"

#include <fml/gpu/card.hh>
#include <fml/gpu/gpumat.hh>
#include <fml/gpu/linalg.hh>

using namespace fml;


template <typename REAL>
static inline void add(bool transx, bool transy, REAL alpha, REAL beta, void *x, void *y, void *ret)
{
  CAST_MAT(gpumat, REAL, x_cast, x);
  CAST_MAT(gpumat, REAL, y_cast, y);
  CAST_MAT(gpumat, REAL, ret_cast, ret);
  linalg::add(transx, transy, alpha, beta, *x_cast, *y_cast, *ret_cast);
}

extern "C" SEXP R_gpumat_linalg_add(SEXP type, SEXP transx, SEXP transy, SEXP alpha, SEXP beta, SEXP x_robj, SEXP y_robj, SEXP ret_robj)
{
  void *x = getRptr(x_robj);
  void *y = getRptr(y_robj);
  void *ret = getRptr(ret_robj);
  APPLY_TEMPLATED_FUNCTION(type, add, LGL(transx), LGL(transy), DBL(alpha), DBL(beta), x, y, ret);
  
  return R_NilValue;
}



template <typename REAL>
static inline void matmult(bool transx, bool transy, REAL alpha, void *x, void *y, void *ret)
{
  CAST_MAT(gpumat, REAL, x_cast, x);
  CAST_MAT(gpumat, REAL, y_cast, y);
  CAST_MAT(gpumat, REAL, ret_cast, ret);
  linalg::matmult(transx, transy, alpha, *x_cast, *y_cast, *ret_cast);
}

extern "C" SEXP R_gpumat_linalg_matmult(SEXP type, SEXP transx, SEXP transy, SEXP alpha, SEXP x_robj, SEXP y_robj, SEXP ret_robj)
{
  void *x = getRptr(x_robj);
  void *y = getRptr(y_robj);
  void *ret = getRptr(ret_robj);
  APPLY_TEMPLATED_FUNCTION(type, matmult, LGL(transx), LGL(transy), DBL(alpha), x, y, ret);
  
  return R_NilValue;
}



template <typename REAL>
static inline void crossprod(bool xpose, REAL alpha, void *x, void *ret)
{
  CAST_MAT(gpumat, REAL, x_cast, x);
  CAST_MAT(gpumat, REAL, ret_cast, ret);
  if (xpose)
    linalg::tcrossprod(alpha, *x_cast, *ret_cast);
  else
    linalg::crossprod(alpha, *x_cast, *ret_cast);
}

extern "C" SEXP R_gpumat_linalg_crossprod(SEXP type, SEXP xpose, SEXP alpha, SEXP x_robj, SEXP ret_robj)
{
  void *x = getRptr(x_robj);
  void *ret = getRptr(ret_robj);
  APPLY_TEMPLATED_FUNCTION(type, crossprod, LGL(xpose), DBL(alpha), x, ret);
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_xpose(SEXP type, SEXP x_robj, SEXP ret_robj)
{
  #define FMLR_TMP_XPOSE(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    CAST_FML(gpumat, type, ret, ret_robj); \
    linalg::xpose(*x, *ret); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_XPOSE, type);
  #undef FMLR_TMP_XPOSE
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_lu(SEXP type, SEXP x_robj)
{
  #define FMLR_TMP_LU(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    linalg::lu(*x); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_LU, type);
  #undef FMLR_TMP_LU
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_det(SEXP type, SEXP x_robj)
{
  SEXP ret, ret_names;
  SEXP sign, modulus;
  
  PROTECT(ret = allocVector(VECSXP, 2));
  PROTECT(ret_names = allocVector(STRSXP, 2));
  
  PROTECT(sign = allocVector(INTSXP, 1));
  PROTECT(modulus = allocVector(REALSXP, 1));
  
  #define FMLR_TMP_DET(type) { \
    type mod; \
    CAST_FML(gpumat, type, x, x_robj); \
    linalg::det(*x, INT(sign), mod); \
    DBL(modulus) = (double) mod; }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_DET, type);
  #undef FMLR_TMP_DET
  
  SET_VECTOR_ELT(ret, 0, sign);
  SET_VECTOR_ELT(ret, 1, modulus);
  SET_STRING_ELT(ret_names, 0, mkChar("sign"));
  SET_STRING_ELT(ret_names, 1, mkChar("modulus"));
  setAttrib(ret, R_NamesSymbol, ret_names);
  
  UNPROTECT(4);
  return ret;
}



extern "C" SEXP R_gpumat_linalg_trace(SEXP type, SEXP x_robj)
{
  SEXP ret;
  PROTECT(ret = allocVector(REALSXP, 1));
  
  #define FMLR_TMP_TRACE(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    DBL(ret) = linalg::trace(*x); }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_TRACE(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_TRACE(float)
  else //if (INT(type) == TYPE_INT)
    FMLR_TMP_TRACE(int)
  
  #undef FMLR_TMP_TRACE
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_linalg_svd(SEXP type, SEXP x_robj, SEXP s_robj, SEXP u_robj, SEXP vt_robj)
{
  #define FMLR_TMP_SVD(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *s = (gpuvec<type>*) getRptr(s_robj); \
    if (u_robj == R_NilValue) \
      linalg::svd(*x, *s); \
    else { \
      gpumat<type> *u = (gpumat<type>*) getRptr(u_robj); \
      gpumat<type> *vt = (gpumat<type>*) getRptr(vt_robj); \
      linalg::svd(*x, *s, *u, *vt); } }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_SVD(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_SVD(float)
  else
    error(TYPE_ERR);
  
  #undef FMLR_TMP_SVD
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_eigen_sym(SEXP type, SEXP x_robj, SEXP values_robj, SEXP vectors_robj)
{
  #define FMLR_TMP_EIGEN_SYM(type) { \
    gpumat<type> *x = (gpumat<type>*) getRptr(x_robj); \
    gpuvec<type> *values = (gpuvec<type>*) getRptr(values_robj); \
    if (vectors_robj == R_NilValue) \
      linalg::eigen_sym(*x, *values); \
    else { \
      gpumat<type> *vectors = (gpumat<type>*) getRptr(vectors_robj); \
      linalg::eigen_sym(*x, *values, *vectors); } }
  
  if (INT(type) == TYPE_DOUBLE)
    FMLR_TMP_EIGEN_SYM(double)
  else if (INT(type) == TYPE_FLOAT)
    FMLR_TMP_EIGEN_SYM(float)
  else
    error(TYPE_ERR);
  
  #undef FMLR_TMP_EIGEN_SYM
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_invert(SEXP type, SEXP x_robj)
{
  #define FMLR_TMP_INVERT(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    linalg::invert(*x); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_INVERT, type);
  #undef FMLR_TMP_INVERT
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_solve(SEXP type, SEXP x_robj, SEXP y_class, SEXP y_robj)
{
  #define FMLR_TMP_SOLVE(type) \
    if (INT(y_class) == CLASS_VEC){ \
      gpuvec<type> *y = (gpuvec<type>*) getRptr(y_robj); \
      linalg::solve(*x, *y); \
    } else { \
      gpumat<type> *y = (gpumat<type>*) getRptr(y_robj); \
      linalg::solve(*x, *y); }
  
  if (INT(type) == TYPE_DOUBLE)
  {
    gpumat<double> *x = (gpumat<double>*) getRptr(x_robj);
    FMLR_TMP_SOLVE(double)
  }
  else if (INT(type) == TYPE_FLOAT)
  {
    gpumat<float> *x = (gpumat<float>*) getRptr(x_robj);
    FMLR_TMP_SOLVE(float)
  }
  else
    error(TYPE_ERR);
  
  #undef FMLR_TMP_SOLVE
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_qr(SEXP type, SEXP x_robj, SEXP qraux_robj)
{
  #define FMLR_TMP_QR(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    CAST_FML(gpuvec, type, qraux, qraux_robj); \
    linalg::qr(false, *x, *qraux); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_QR, type);
  #undef FMLR_TMP_QR
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_qr_Q(SEXP type, SEXP QR_robj, SEXP qraux_robj, SEXP Q_robj, SEXP work_robj)
{
  #define FMLR_TMP_QR_Q(type) { \
    CAST_FML(gpumat, type, QR, QR_robj); \
    CAST_FML(gpuvec, type, qraux, qraux_robj); \
    CAST_FML(gpumat, type, Q, Q_robj); \
    CAST_FML(gpuvec, type, work, work_robj); \
    linalg::qr_Q(*QR, *qraux, *Q, *work); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_QR_Q, type);
  #undef FMLR_TMP_QR_Q
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_qr_R(SEXP type, SEXP QR_robj, SEXP R_robj)
{
  #define FMLR_TMP_QR_R(type) { \
    CAST_FML(gpumat, type, QR, QR_robj); \
    CAST_FML(gpumat, type, R, R_robj); \
    linalg::qr_R(*QR, *R); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_QR_R, type);
  #undef FMLR_TMP_QR_R
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_lq(SEXP type, SEXP x_robj, SEXP lqaux_robj)
{
  #define FMLR_TMP_LQ(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    CAST_FML(gpuvec, type, lqaux, lqaux_robj); \
    linalg::lq(*x, *lqaux); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_LQ, type);
  #undef FMLR_TMP_LQ
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_lq_L(SEXP type, SEXP LQ_robj, SEXP L_robj)
{
  #define FMLR_TMP_LQ_L(type) { \
    CAST_FML(gpumat, type, LQ, LQ_robj); \
    CAST_FML(gpumat, type, L, L_robj); \
    linalg::lq_L(*LQ, *L); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_LQ_L, type);
  #undef FMLR_TMP_LQ_L
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_lq_Q(SEXP type, SEXP LQ_robj, SEXP lqaux_robj, SEXP Q_robj, SEXP work_robj)
{
  #define FMLR_TMP_LQ_Q(type) { \
    CAST_FML(gpumat, type, LQ, LQ_robj); \
    CAST_FML(gpuvec, type, lqaux, lqaux_robj); \
    CAST_FML(gpumat, type, Q, Q_robj); \
    CAST_FML(gpuvec, type, work, work_robj); \
    linalg::lq_Q(*LQ, *lqaux, *Q, *work); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_LQ_Q, type);
  #undef FMLR_TMP_LQ_Q
  
  return R_NilValue;
}


extern "C" SEXP R_gpumat_linalg_tssvd(SEXP type, SEXP x_robj, SEXP s_robj, SEXP u_robj, SEXP vt_robj)
{
  #define FMLR_TMP_TSSVD(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    CAST_FML(gpuvec, type, s, s_robj); \
    if (u_robj == R_NilValue) \
      linalg::svd(*x, *s); \
    else { \
      CAST_FML(gpumat, type, u, u_robj); \
      CAST_FML(gpumat, type, vt, vt_robj); \
      linalg::tssvd(*x, *s, *u, *vt); } }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_TSSVD, type);
  #undef FMLR_TMP_TSSVD
  
  return R_NilValue;
}


extern "C" SEXP R_gpumat_linalg_cpsvd(SEXP type, SEXP x_robj, SEXP s_robj, SEXP u_robj, SEXP vt_robj)
{
  #define FMLR_TMP_CPSVD(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    CAST_FML(gpuvec, type, s, s_robj); \
    if (u_robj == R_NilValue) \
      linalg::svd(*x, *s); \
    else { \
      CAST_FML(gpumat, type, u, u_robj); \
      CAST_FML(gpumat, type, vt, vt_robj); \
      linalg::cpsvd(*x, *s, *u, *vt); } }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_CPSVD, type);
  #undef FMLR_TMP_CPSVD
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_chol(SEXP type, SEXP x_robj)
{
  #define FMLR_TMP_CHOL(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    linalg::chol(*x); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_CHOL, type);
  #undef FMLR_TMP_CHOL
  
  return R_NilValue;
}



extern "C" SEXP R_gpumat_linalg_norm(SEXP type, SEXP x_robj, SEXP norm)
{
  SEXP ret;
  
  PROTECT(ret = allocVector(REALSXP, 1));
  
  #define FMLR_TMP_NORM(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    if (CHR(norm) == '1') \
      DBL(ret) = (double)linalg::norm_1(*x); \
    else if (CHR(norm) == 'I') \
      DBL(ret) = (double)linalg::norm_I(*x); \
    else if (CHR(norm) == 'F') \
      DBL(ret) = (double)linalg::norm_F(*x); \
    else if (CHR(norm) == 'M') \
      DBL(ret) = (double)linalg::norm_M(*x); \
    else /*if (CHR(norm) == '2')*/ \
      DBL(ret) = (double)linalg::norm_2(*x); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_NORM, type);
  #undef FMLR_TMP_NORM
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_linalg_cond(SEXP type, SEXP x_robj, SEXP norm)
{
  SEXP ret;
  
  PROTECT(ret = allocVector(REALSXP, 1));
  
  #define FMLR_TMP_COND(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    if (CHR(norm) == '1') \
      DBL(ret) = (double)linalg::cond_1(*x); \
    else if (CHR(norm) == 'I') \
      DBL(ret) = (double)linalg::cond_I(*x); \
    else /*if (CHR(norm) == '2')*/ \
      DBL(ret) = (double)linalg::cond_2(*x); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_COND, type);
  #undef FMLR_TMP_COND
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_linalg_dot(SEXP type, SEXP x_robj, SEXP y_robj)
{
  SEXP ret;
  
  PROTECT(ret = allocVector(REALSXP, 1));
  
  #define FMLR_TMP_DOT(type) { \
    CAST_FML(gpuvec, type, x, x_robj); \
    if (y_robj == R_NilValue) \
      DBL(ret) = (double)linalg::dot(*x); \
    else { \
      CAST_FML(gpuvec, type, y, y_robj); \
      DBL(ret) = (double)linalg::dot(*x, *y); } }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_DOT, type);
  #undef FMLR_TMP_DOT
  
  UNPROTECT(1);
  return ret;
}



extern "C" SEXP R_gpumat_linalg_trinv(SEXP type, SEXP upper, SEXP unit_diag, SEXP x_robj)
{
  #define FMLR_TMP_TRINV(type) { \
    CAST_FML(gpumat, type, x, x_robj); \
    linalg::trinv(INT(upper), INT(unit_diag), *x); }
  
  APPLY_TEMPLATED_MACRO(FMLR_TMP_TRINV, type);
  #undef FMLR_TMP_TRINV
  
  return R_NilValue;
}
